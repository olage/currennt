#include "hip/hip_runtime.h"
#include "CtcPostOutputLayer.hpp"
#include "../helpers/getRawPointer.cuh"
#include "../helpers/NumericLimits.cuh"
#include "../helpers/safeExp.cuh"
#include <iostream>

#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>

class LogScale {
    public:
        __host__ __device__ inline static real_t sum(real_t a, real_t b) {
            if (a < b) {
                real_t c = a; a = b; b = c;
            }
            real_t e = b - a;
            if (e > -helpers::NumericLimits<real_t>::expLimit()) {
                a += std::log(1 + exp(e));
            }
            return a;
        }
        __host__ __device__ inline static real_t sub(real_t a, real_t b) {
            if (a < b) {
                return helpers::NumericLimits<real_t>::logZero();
            }


            real_t e = b - a;
            if (e > -helpers::NumericLimits<real_t>::expLimit()) {
                a += log(1 - exp(e));
            }


            return a;
        }


        __host__ __device__ inline static real_t mult(real_t a, real_t b) {
            return a + b;
        }


        __host__ __device__ inline static real_t div(real_t a, real_t b) {
            return a - b;
        }


        __host__ __device__ inline static real_t linear_log(real_t a) {
            return a;
        }


        struct LinearLog {
            __host__ __device__ real_t operator()(const real_t& a) const { return a; }
        };


        __host__ __device__ inline static real_t to_scale(real_t a) {
            if (a <= helpers::NumericLimits<real_t>::min()) {
                return helpers::NumericLimits<real_t>::logZero();
            }
            else {
                return log(a);
            }
        }


        __host__ __device__ inline static real_t to_linear(real_t a) {
            return helpers::safeExp(a);
        }


        __host__ __device__ inline static real_t one() {
            return 0;
        }


        __host__ __device__ inline static real_t zero() {
            return helpers::NumericLimits<real_t>::logZero();
        }
};


typedef LogScale sc;



namespace internal {
    namespace {
        struct ComputeAlphaFn
        {
            public:
                const real_t *prevAlphas;
                const real_t *inputs;
                const real_t *setAlpha;


                const char *patTypes;


                const int *fmap;
                const int *imap;


                int maxWordLen;
                int parallelSequences;


                __host__ __device__ real_t operator() (const int &outputIdx)
                {
                    int patIdx = outputIdx / maxWordLen;
                    int blockIdx = outputIdx % (maxWordLen * parallelSequences);


                    if (patTypes[patIdx] == PATTYPE_NONE) {
                        return sc::zero();
                    }


                    if (patTypes[patIdx] == PATTYPE_FIRST) {
                        return sc::mult(setAlpha[blockIdx], sc::to_scale(inputs[imap[blockIdx]]));
                    }


                    real_t output = sc::zero();


                    for (int j = fmap[blockIdx]; j <= blockIdx; ++j) {
                        output = sc::sum(output, prevAlphas[j]);
                    }


                    output = sc::mult(output, sc::to_scale(inputs[imap[blockIdx]]));


                    return output;
                }
        };


        struct ComputeBetaFn
        {
            public:
                const real_t *nextBeta;
                const real_t *inputs;
                const real_t *setBeta;
                const char *patTypes;


                const int *gmap;
                const int *imap;


                int maxWordLen;
                int parallelSequences;


                __host__ __device__ real_t operator() (const int &outputIdx)
                {
                    int patIdx = outputIdx / maxWordLen;
                    int blockIdx = outputIdx % (maxWordLen * parallelSequences);


                    if (patTypes[patIdx] == PATTYPE_LAST) {
                        return setBeta[blockIdx];
                    }


                    if (patTypes[patIdx] == PATTYPE_NONE) {
                        return sc::zero();
                    }


                    real_t output = sc::zero();


                    for (int j = blockIdx; j <= gmap[blockIdx]; ++j) {
                        real_t s = sc::mult(nextBeta[j], sc::to_scale(inputs[imap[j]]));
                        output = sc::sum(output, s);
                    }


                    return output;
                }
        };


        struct ComputeWordErrorFn
        {
            const real_t *alphas;
            const real_t *betas;
            int maxWordLen;


            public:
            __host__ __device__ real_t operator() (const int &outputIdx)
            {
                int i = outputIdx * maxWordLen;
                return sc::sum(sc::mult(betas[i], alphas[i]), sc::mult(betas[i + 1], alphas[i + 1]));
            }
        };


        struct ComputeOutputErrorPhaseFn
        {
            const real_t *alphas;
            const real_t *betas;
            const real_t *wordErrors;
            const char *patTypes;


            const int *imap;
            int maxWordLen;
            int parallelSequences;
            int prevLayerSize;


            public:
            __host__ __device__ real_t operator() (const real_t &y, const int &outputIdx)
            {
                int seqIdx = (outputIdx / prevLayerSize) % parallelSequences;
                int blockIdx = outputIdx % (prevLayerSize * parallelSequences);
                int patIdx = outputIdx / prevLayerSize;
                if (patTypes[patIdx] == PATTYPE_NONE) {
                    return 0.f;
                }
                real_t output = sc::zero();
                real_t num = sc::zero();
                real_t den = sc::zero();


                for (int i = 0; i < maxWordLen; ++i) {
                    real_t cur = sc::mult(alphas[patIdx * maxWordLen + i], betas[patIdx * maxWordLen + i]);
                    if (imap[seqIdx * maxWordLen + i] == blockIdx) {
                        num = sc::sum(num, cur);
                    }
                    den = sc::sum(den, cur);
                }
                real_t error = -sc::to_linear(sc::div(num, den)) / y;
                return error;
            }
        };
    }
}


namespace layers {


    template <typename TDevice>
        CtcPostOutputLayer<TDevice>::CtcPostOutputLayer(const helpers::JsonValue &layerChild, Layer<TDevice> &precedingLayer)
        : PostOutputLayer<TDevice>(layerChild, precedingLayer, 1), prevLayerSize(precedingLayer.size())
        {
        }


    template <typename TDevice>
        CtcPostOutputLayer<TDevice>::~CtcPostOutputLayer()
        {
        }


    template <typename TDevice>
        const std::string& CtcPostOutputLayer<TDevice>::type() const
        {
            static const std::string s("ctc");
            return s;
        }

    template <typename TDevice>
        void CtcPostOutputLayer<TDevice>::findAlphas()
        {
            m_alphas.resize(maxWordLen * this->parallelSequences() * this->curMaxSeqLength());


            real_vector &y = this->_actualOutputs();


            internal::ComputeAlphaFn fn;
            fn.prevAlphas = helpers::getRawPointer(m_alphas);
            fn.fmap = helpers::getRawPointer(fmap);
            fn.imap = helpers::getRawPointer(imap);
            fn.inputs = helpers::getRawPointer(this->_actualOutputs());
            fn.patTypes = helpers::getRawPointer(this->patTypes());
            fn.setAlpha = helpers::getRawPointer(this->setAlpha);
            fn.maxWordLen = this->maxWordLen;
            fn.parallelSequences = this->parallelSequences();


            int alphaTimeSize = this->maxWordLen * this->parallelSequences();
            int inputTimeSize = this->prevLayerSize * this->parallelSequences();


            for (int t = 0; t < this->curMaxSeqLength(); ++t) {


                thrust::transform(
                        thrust::counting_iterator<int>(t*alphaTimeSize),
                        thrust::counting_iterator<int>(t*alphaTimeSize) + alphaTimeSize,
                        m_alphas.begin() + t*alphaTimeSize,
                        fn
                        );


                fn.inputs += inputTimeSize;
                if(t != 0) fn.prevAlphas += alphaTimeSize;
            }
        }


    template <typename TDevice>
        void CtcPostOutputLayer<TDevice>::findBetas()
        {
            m_betas.resize(maxWordLen * this->parallelSequences() * this->curMaxSeqLength());


            internal::ComputeBetaFn fn;
            fn.nextBeta = helpers::getRawPointer(m_betas) + m_betas.size();
            fn.gmap = helpers::getRawPointer(gmap);
            fn.imap = helpers::getRawPointer(imap);
            fn.inputs = helpers::getRawPointer(this->_actualOutputs()) + this->prevLayerSize * this->parallelSequences() * this->curMaxSeqLength();
            fn.patTypes = helpers::getRawPointer(this->patTypes());
            fn.setBeta = helpers::getRawPointer(this->setBeta);
            fn.maxWordLen = this->maxWordLen;
            fn.parallelSequences = this->parallelSequences();


            int betaTimeSize = maxWordLen * this->parallelSequences();
            int inputTimeSize = prevLayerSize * this->parallelSequences();


            for (int t = this->curMaxSeqLength() - 1; t >= 0; --t) {
                thrust::transform(
                        thrust::counting_iterator<int>(t*betaTimeSize),
                        thrust::counting_iterator<int>(t*betaTimeSize) + betaTimeSize,
                        m_betas.begin() + t*betaTimeSize,
                        fn
                        );


                fn.inputs -= inputTimeSize;
                fn.nextBeta -= betaTimeSize;


            }
        }

    template <typename TDevice>
        real_t CtcPostOutputLayer<TDevice>::calculateError()
        {
            wordErrors.resize(this->curNumSeqs());


            internal::ComputeWordErrorFn fn;
            fn.alphas = helpers::getRawPointer(m_alphas);
            fn.betas = helpers::getRawPointer(m_betas);
            fn.maxWordLen = this->maxWordLen;


            thrust::transform(
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(this->curNumSeqs()),
                    wordErrors.begin(),
                    fn
                    );
            real_t error = -(thrust::transform_reduce(wordErrors.begin(), wordErrors.end(), sc::LinearLog(), 0.f, thrust::plus<real_t>()));
            return error;
        }


    template <typename TDevice>
        void CtcPostOutputLayer<TDevice>::computeForwardPass()
        {
            findAlphas();
            findBetas();
        }


    template <typename TDevice>
        void CtcPostOutputLayer<TDevice>::computeBackwardPass()
        {
            real_vector &outputErrors = this->_outputErrors();
            real_vector &input = this->_actualOutputs();


            internal::ComputeOutputErrorPhaseFn fn;
            fn.alphas = helpers::getRawPointer(m_alphas);
            fn.betas = helpers::getRawPointer(m_betas);
            fn.wordErrors = helpers::getRawPointer(wordErrors);
            fn.imap = helpers::getRawPointer(imap);
            fn.patTypes = helpers::getRawPointer(this->patTypes());
            fn.maxWordLen = this->maxWordLen;
            fn.prevLayerSize = this->prevLayerSize;
            fn.parallelSequences = this->curNumSeqs();


            thrust::transform(input.begin(), input.end(), thrust::counting_iterator<int>(0), outputErrors.begin(), fn);
        }


    template <typename TDevice>
        void CtcPostOutputLayer<TDevice>::loadSequences(const data_sets::DataSetFraction &fraction)
        {
            PostOutputLayer<TDevice>::loadSequences(fraction);


            real_vector &o = this->_targets();
            int n = this->curMaxSeqLength();


            std::vector< std::vector<int> > words(this->curNumSeqs());


            for (size_t i = 0; i < this->curNumSeqs(); ++i) {
                for (int j = i; int(o[j] + 0.5) != 0; j += this->curNumSeqs()) {
                    words[i].push_back(int(o[j] + 0.5));
                }
            }


            this->maxWordLen = 0;
            for(size_t i = 0; i < words.size(); ++i) {
                this->maxWordLen = std::max(this->maxWordLen, 2 * words[i].size() + 1);
            }


            this->computationSize = this->curNumSeqs() * this->maxWordLen;


            imap.resize(this->computationSize);


            setBeta = real_vector(this->computationSize, sc::zero());
            setAlpha = real_vector(this->computationSize, sc::zero());


            for (int i = 0; i < this->curNumSeqs(); ++i) {
                int word_offset = i * this->maxWordLen;
                int input_offset = i * this->prevLayerSize;


                std::fill(imap.begin() + word_offset, imap.begin() + word_offset + maxWordLen, input_offset);


                for (size_t j = 0; j < words[i].size(); ++j) {
                    imap[word_offset + 2 * j + 1] += words[i][j];
                }


                setAlpha[word_offset] = sc::one();
                setAlpha[word_offset + 1] = sc::one();


                setBeta[word_offset + 2 * words[i].size()] = sc::one();
                setBeta[word_offset + 2 * words[i].size() - 1] = sc::one();
            }


            fmap.resize(this->maxWordLen * this->curNumSeqs(), this->maxWordLen * this->curNumSeqs());
            gmap.resize(this->maxWordLen * this->curNumSeqs(), 0);


            for (int i = 0; i < this->curNumSeqs(); ++i) {
                int longWordSize = 2 * words[i].size() + 1;
                for (int u = 0; u < longWordSize; ++u) {
                    int let = (u % 2 == 0) ? 0 : (u / 2);
                    if (u % 2 == 0 || (let >= 1 && words[i][let] == words[i][let - 1])) {
                        fmap[i * maxWordLen + u] = std::max(0, u - 1) + i * maxWordLen;
                    }
                    else {
                        fmap[i * maxWordLen + u] = std::max(0, u - 2) + i * maxWordLen;
                    }


                    if (u % 2 == 0 || ((let + 1 < words[i].size()) && (words[i][let + 1] == words[i][let]))) {
                        gmap[i * maxWordLen + u] = std::min(longWordSize - 1, u + 1) + i * maxWordLen;
                    }
                    else {
                        gmap[i * maxWordLen + u] = std::min(longWordSize - 1, u + 2) + i * maxWordLen;
                    }
                }
            }

        }


    template class CtcPostOutputLayer<Cpu>;
    template class CtcPostOutputLayer<Gpu>;
};
